
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void _sobel_process_kernel_(unsigned char* d_src, unsigned char* d_dst, int row, int col)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int src_center = idy * col + idx;
    if(idy >= row || idx >= col)
        return;
    int lidx = (idx <= 0)? 0 : idx - 1;
    int ridx = (idx >= col - 1)? col - 1 : idx + 1;
    int uidy = (idy <= 0)? 0 : idy - 1;
    int didy = (idy >= row - 1)? row - 1 : idy + 1;
    int src_left = idy * col + lidx;
    int src_right = idy * col + ridx;
    int src_up = uidy * col + idx;
    int src_up_left = uidy * col + lidx; 
    int src_up_right = uidy * col + ridx;
    int src_down = didy * col + idx;
    int src_down_left = didy * col + lidx;
    int src_down_right = didy * col + ridx;
    float src_left_r = d_src[src_left];
    float src_right_r = d_src[src_right];
    float src_up_r = d_src[src_up];
    float src_up_left_r = d_src[src_up_left];
    float src_up_right_r = d_src[src_up_right];
    float src_down_r = d_src[src_down];
    float src_down_left_r = d_src[src_down_left];
    float src_down_right_r = d_src[src_down_right];
    float GX = 1 * src_up_right_r + 2 * src_right_r + 1 * src_down_right_r - 1 * src_up_left_r - 2 * src_left_r - 1 * src_down_left_r;
    float GY = 1 * src_up_left_r + 2 * src_up_r + 1 * src_up_right_r - 1 * src_down_left_r - 2 * src_down_r - 1 * src_down_right_r;
    float G = sqrt(pow(GX, 2) + pow(GY, 2));
    unsigned char gray = static_cast<unsigned char>(G);
    d_dst[src_center] = gray;
}

__global__ void _split_channel_kernel_(unsigned char* d_src, unsigned char* d_r, unsigned char* d_g, unsigned char* d_b, int row, int col)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int id = idy * col + idx;
    if(id >= row * col)
        return;
    d_r[id] = d_src[id * 3 + 0];
    d_g[id] = d_src[id * 3 + 1];
    d_b[id] = d_src[id * 3 + 2];
}

extern void _sobel_process_(unsigned char* src, unsigned char* dst, int row, int col)
{
    unsigned char* d_src = nullptr;
    unsigned char* d_dst = nullptr;
    const size_t ARRAY_BYTES = row * col * sizeof(unsigned char);
    hipMalloc((void**) &d_src, ARRAY_BYTES);
    hipMalloc((void**) &d_dst, ARRAY_BYTES);
    hipMemcpy(d_src, src, ARRAY_BYTES, hipMemcpyHostToDevice);
    dim3 threads(32, 32);
    dim3 blocks(col / threads.x + 1, row / threads.y + 1);
    _sobel_process_kernel_<<<blocks, threads>>>(d_src, d_dst, row, col);
    hipDeviceSynchronize();
    hipMemcpy(dst, d_dst, ARRAY_BYTES, hipMemcpyDeviceToHost);
    hipFree(d_src);
    hipFree(d_dst);
}

extern void _split_channel_(unsigned char* src, unsigned char* r, unsigned char* g, unsigned char* b, int row, int col)
{
    unsigned char* d_src = nullptr;
    unsigned char* d_r = nullptr;
    unsigned char* d_g = nullptr;
    unsigned char* d_b = nullptr;
    const size_t ARRAY_BYTES = row * col * sizeof(unsigned char);
    hipMalloc((void**) &d_src, ARRAY_BYTES * 3);
    hipMalloc((void**) &d_r, ARRAY_BYTES);
    hipMalloc((void**) &d_g, ARRAY_BYTES);
    hipMalloc((void**) &d_b, ARRAY_BYTES);
    hipMemcpy(d_src, src, ARRAY_BYTES * 3, hipMemcpyHostToDevice);
    dim3 threads(1, 1);
    dim3 blocks(col / threads.x + 1, row / threads.y + 1);
    _split_channel_kernel_<<<blocks, threads>>>(d_src, d_r, d_g, d_b, row, col);
    hipDeviceSynchronize();
    hipMemcpy(r, d_r, ARRAY_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(g, d_g, ARRAY_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, ARRAY_BYTES, hipMemcpyDeviceToHost);
    hipFree(d_src);
    hipFree(d_r);
    hipFree(d_g);
    hipFree(d_b);
}